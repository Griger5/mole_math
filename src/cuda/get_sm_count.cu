
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void) {
    int deviceId;
    int num_of_SM;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&num_of_SM, hipDeviceAttributeMultiprocessorCount, deviceId);

    printf("%d", num_of_SM);

    return 0;
}