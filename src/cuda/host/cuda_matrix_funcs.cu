#include "../../../include/mole_math/cuda_matrix_funcs.cuh"

#include "../device/cuda_kernel_matrix_funcs.cuh"

#include "../../../include/mole_math/cuda_check_error.cuh"

const int threads_per_block = 512;

double cuda_matrix_sum_row(const Matrix matrix, size_t row) {
    int deviceId;
    int num_of_SM;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&num_of_SM, hipDeviceAttributeMultiprocessorCount, deviceId);

    const int blocks_per_grid = 4 * num_of_SM;
    
    size_t rows = matrix.rows;
    size_t cols = matrix.cols;
    size_t row_size_bytes = cols * sizeof(double);

    if (row >= rows) return NAN;

    double *d_row_values;
    double *d_sum_blocks, *d_sum;
    double h_sum;
    
    checkCuda( hipMalloc(&d_row_values, row_size_bytes) );
    checkCuda( hipMalloc(&d_sum_blocks, blocks_per_grid * sizeof(double)) );
    checkCuda( hipMalloc(&d_sum, sizeof(double)) );

    checkCuda( hipMemcpy(d_row_values, matrix.values[row], row_size_bytes, hipMemcpyHostToDevice) );

    cuda_kernel_matrix_sum_row<<<blocks_per_grid, threads_per_block, threads_per_block*sizeof(double)>>>(d_row_values, cols, d_sum_blocks);
    checkCuda(hipGetLastError());

    cuda_kernel_matrix_sum_row<<<1, 2*threads_per_block, 2*threads_per_block*sizeof(double)>>>(d_sum_blocks, blocks_per_grid, d_sum);
    checkCuda(hipGetLastError());

    checkCuda( hipMemcpy(&h_sum, d_sum, sizeof(double), hipMemcpyDeviceToHost) );

    hipFree(d_row_values);
    hipFree(d_sum_blocks);
    hipFree(d_sum);

    return h_sum;
}